#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/l2n_layer.hpp"

#include "thrust/device_vector.h"

namespace caffe {

template <typename Dtype>
    __global__ void kernel_channel_sum(const int num, const int channels,
                const int spatial_dim, const Dtype* data, Dtype* channel_sum) {
      CUDA_KERNEL_LOOP(index, num * spatial_dim) {
          int n = index / spatial_dim;
          int s = index % spatial_dim;
          Dtype sum = 0;
          for (int c = 0; c < channels; ++c) {
                sum += data[(n * channels + c) * spatial_dim + s];
              }
          channel_sum[index] = sum;
        }
    }

template <typename Dtype>
    __global__ void kernel_channel_mul(const int num, const int channels,
                const int spatial_dim, Dtype* data, const Dtype* channel_sum) {
      CUDA_KERNEL_LOOP(index, num * spatial_dim) {
          int n = index / spatial_dim;
          int s = index % spatial_dim;
          for (int c = 0; c < channels; ++c) {
                data[(n * channels + c) * spatial_dim + s] *= channel_sum[index];
              }
        }
    }

template <typename Dtype>
    __global__ void kernel_channel_div(const int num, const int channels,
                const int spatial_dim, Dtype* data, const Dtype* channel_sum) {
      CUDA_KERNEL_LOOP(index, num * spatial_dim) {
          int n = index / spatial_dim;
          int s = index % spatial_dim;
          for (int c = 0; c < channels; ++c) {
                data[(n * channels + c) * spatial_dim + s] /= channel_sum[index];
              }
        }
    }

template <typename Dtype>
    __global__ void kernel_channel_dot(const int num, const int channels,
                const int spatial_dim, const Dtype* data_1, const Dtype* data_2,
                    Dtype* channel_dot) {
      CUDA_KERNEL_LOOP(index, num * spatial_dim) {
          int n = index / spatial_dim;
          int s = index % spatial_dim;
          Dtype dot = 0;
          for (int c = 0; c < channels; ++c) {
                dot += (data_1[(n * channels + c) * spatial_dim + s]
                                  * data_2[(n * channels + c) * spatial_dim + s]);
              }
          channel_dot[index] = dot;
        }
    }

template <typename Dtype>
    void L2NLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
              const  vector<Blob<Dtype>*>& top) {
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* top_data = top[0]->mutable_gpu_data();
      Dtype* square_data = square_.mutable_gpu_data();
      Dtype* norm_data = norm_.mutable_gpu_data();
      int num = bottom[0]->num();
      int channels = bottom[0]->channels();
      int spatial_dim = bottom[0]->height() * bottom[0]->width();
      caffe_copy(bottom[0]->count(), bottom_data, top_data);
      caffe_copy(bottom[0]->count(), bottom_data, square_data);
    
      // square
      caffe_gpu_powx<Dtype>(bottom[0]->count(), square_data, Dtype(2.0), square_data);
      //sum cross channel
      kernel_channel_sum<Dtype><<<CAFFE_GET_BLOCKS(num * spatial_dim),
          CAFFE_CUDA_NUM_THREADS>>>(num, channels, spatial_dim, square_data,
                        norm_data);
      // square root
      caffe_gpu_powx<Dtype>(num * spatial_dim, norm_data, Dtype(0.5), norm_data);
      // divide
      kernel_channel_div<Dtype><<<CAFFE_GET_BLOCKS(num * spatial_dim),
          CAFFE_CUDA_NUM_THREADS>>>(num, channels, spatial_dim, top_data,
                        norm_data);
    }

template <typename Dtype>
    void L2NLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                const vector<bool>& propagate_down,
                const vector<Blob<Dtype>*>& bottom) {
      const Dtype* top_diff = top[0]->gpu_diff();
      const Dtype* top_data = top[0]->gpu_data();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* norm_data = norm_.mutable_gpu_data();
      Dtype* temp_dot_data = temp_dot_.mutable_gpu_data();
      Dtype* temp_data = square_.mutable_gpu_data();//just reuse the square_
      int num = top[0]->num();
      int channels = top[0]->channels();
      int spatial_dim = top[0]->height() * top[0]->width();
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
      caffe_copy(top[0]->count(), bottom_data, temp_data);
    
      // b_diff = t_diff / norm - dot(t_diff, t_data) / (norm)^2 * bottom_data
      // temp_dot_data = dot(t_diff, t_data)
      kernel_channel_dot<Dtype><<<CAFFE_GET_BLOCKS(num * spatial_dim),
          CAFFE_CUDA_NUM_THREADS>>>(num, channels, spatial_dim, top_diff, top_data,
                        temp_dot_data);
      // temp_dot_data /= (norm)^2
      caffe_gpu_div<Dtype>(num * spatial_dim, temp_dot_data, norm_data, temp_dot_data);
      caffe_gpu_div<Dtype>(num * spatial_dim, temp_dot_data, norm_data, temp_dot_data);
      // bottom_diff = top_diff, bottom_diff /= norm
      kernel_channel_div<Dtype><<<CAFFE_GET_BLOCKS(num * spatial_dim),
          CAFFE_CUDA_NUM_THREADS>>>(num, channels, spatial_dim, bottom_diff,
                        norm_data);
      // temp_data = bottom_data, temp_data *= temp_dot_data
      kernel_channel_mul<Dtype><<<CAFFE_GET_BLOCKS(num * spatial_dim),
          CAFFE_CUDA_NUM_THREADS>>>(num, channels, spatial_dim, temp_data,
                        temp_dot_data); 
      // bottom_diff += -temp_data
      caffe_gpu_axpy<Dtype>(top[0]->count(), Dtype(-1.0), temp_data, 
                    bottom_diff);
    }

//INSTANTIATE_CLASS(L2NLayer);

INSTANTIATE_LAYER_GPU_FUNCS(L2NLayer);

}  // namespace caffe
